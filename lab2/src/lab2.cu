#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call) 														\
	do	{																\
		hipError_t status = call;										\
		if (status != hipSuccess) {									\
			fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__,		\
								hipGetErrorString(status));			\
			exit(0);													\
		}																\
	} while(0)


texture<uchar4, 2, hipReadModeElementType> rgb_texture;


__device__ double calculate_grad(int x, int y, int* mask)
{
	uchar4 rgb_format;
	double bright;
	double grad = 0;
	int indexing_arr[3] = {-1,0,1};
	int size = 9;
	
	for(int i = 0; i < size; i++)
	{
		rgb_format = tex2D(rgb_texture, x + indexing_arr[i%3], y - indexing_arr[i/3]);
		bright = 0.299 * rgb_format.x + 0.587 * rgb_format.y + 0.114 * rgb_format.z;
		grad += bright * (double)mask[8 - i];
	}

	return grad;
}

__global__ void kernel(uchar4 *output, int h, int w, int* prewitt_mask_x, int* prewitt_mask_y)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	int offset_x = blockDim.x * gridDim.x;
	int offset_y = blockDim.y * gridDim.y;

    uchar4 pixel;	

	for (int x = idx; x < w; x += offset_x)
	{
		for (int y = idy; y < h; y += offset_y)
		{
            pixel = tex2D(rgb_texture, x, y);
			double grad_x = calculate_grad(x, y, prewitt_mask_x);
			double grad_y = calculate_grad(x, y, prewitt_mask_y);
			double total = sqrt(grad_x * grad_x + grad_y * grad_y);

			if(total > UCHAR_MAX)
			{ 
				total = UCHAR_MAX;
			}

			output[y * w + x] = make_uchar4(total, total, total, pixel.w);
		}
	}
}


int main()
{
	int w, h;
	char path_to_input_file[255];
	char path_to_output_file[255];
	scanf("%s",path_to_input_file);
	scanf("%s",path_to_output_file);

	FILE* input_file = fopen(path_to_input_file, "rb");
	fread(&w, sizeof(int), 1, input_file);
	fread(&h, sizeof(int), 1, input_file);
	uchar4* img = (uchar4*)malloc(sizeof(uchar4) * w* h);
	fread(img, sizeof(uchar4), w * h, input_file);
	fclose(input_file);

	int size_mask = 9;
	int prewitt_mask_x[size_mask] = {-1, 0, 1, -1, 0, 1, -1, 0, 1};
	int prewitt_mask_y[size_mask] = {-1, -1, -1, 0, 0, 0, 1, 1, 1};
	
	hipArray *dev_img;
	
	// привязка изображения к uchar4
	hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();

	rgb_texture.addressMode[0] = hipAddressModeClamp;
	rgb_texture.addressMode[1] = hipAddressModeClamp;
	rgb_texture.channelDesc = channel_desc;
	rgb_texture.filterMode  = hipFilterModePoint;
	rgb_texture.normalized  = false;

	int *dev_prewitt_mask_x;
	int *dev_prewitt_mask_y;

	CSC(hipMalloc(&dev_prewitt_mask_x, sizeof(int) *size_mask));
	CSC(hipMalloc(&dev_prewitt_mask_y, sizeof(int) *size_mask));
	CSC(hipMemcpy(dev_prewitt_mask_x, prewitt_mask_x, size_mask*sizeof(int), hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_prewitt_mask_y, prewitt_mask_y, size_mask*sizeof(int), hipMemcpyHostToDevice));
	CSC(hipMallocArray(&dev_img, &channel_desc, w,h));
	CSC(hipMemcpyToArray(dev_img, 0, 0, img, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));
	CSC(hipBindTextureToArray(rgb_texture, dev_img, channel_desc));

	uchar4* dev_output_img;
	CSC(hipMalloc(&dev_output_img, sizeof(uchar4) * w * h));

	hipEvent_t start, stop;
	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

	kernel<<<512, 512>>>(dev_output_img, h, w, dev_prewitt_mask_x, dev_prewitt_mask_y);

	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float t;
    hipEventElapsedTime(&t, start, stop);
	printf("time: %lf\n\n", t);
	hipEventDestroy(start);
    hipEventDestroy(stop);
	
	CSC(hipGetLastError());
	CSC(hipMemcpy(img, dev_output_img, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
	CSC(hipUnbindTexture(rgb_texture));

	FILE* output_file = fopen(path_to_output_file, "wb");

	fwrite(&w, sizeof(int), 1, output_file);
	fwrite(&h, sizeof(int), 1, output_file);
	fwrite(img, sizeof(uchar4), w * h, output_file);
	fclose(output_file);

	CSC(hipFreeArray(dev_img));
	CSC(hipFree(dev_output_img));
	CSC(hipFree(dev_prewitt_mask_x));
	CSC(hipFree(dev_prewitt_mask_y));
	free(img);

	return 0;
}
