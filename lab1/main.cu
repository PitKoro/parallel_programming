
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(int* first_vector, int* second_vector, int* result_vector, int vector_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while(idx < vector_size) {
        result_vector[idx] = first_vector[idx] * second_vector[idx];
        idx += offset;
    }
}

int main() {
    int vector_size = 3;
    int first_vector[3] = {1, 2, 3};
    int second_vector[3] = {4, 5, 6};
    int* result = new int[vector_size];

    int* device_first_vector = new int[vector_size];
    int* device_second_vector = new int[vector_size];
    int* device_result_vector = new int[vector_size];

    hipMalloc(&device_first_vector, sizeof(int) * vector_size);
    hipMalloc(&device_second_vector, sizeof(int) * vector_size);
    hipMalloc(&device_result_vector, sizeof(int) * vector_size);

    hipMemcpy(device_first_vector, &first_vector, sizeof(int) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(device_second_vector, &second_vector, sizeof(int) * vector_size, hipMemcpyHostToDevice);

    kernel<<<256, 256>>>(device_first_vector, device_second_vector, device_result_vector, vector_size);

    hipDeviceSynchronize();
    hipGetLastError();

    hipMemcpy(result, device_result_vector, sizeof(int) * vector_size, hipMemcpyDeviceToHost);

    hipFree(device_first_vector);
    hipFree(device_second_vector);
    hipFree(device_result_vector);

    for (int i = 0; i < vector_size; i++) {
        printf("%d ", result[i]);
    }

    printf("\n");

    return 0;

}